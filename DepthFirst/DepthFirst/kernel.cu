#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <utility>

hipError_t dftrain(const int *data, unsigned int features, unsigned int samples);
int *data,*target,*mask;
std::pair<int, int> feature_threshold[100];
float ft_impurity[100];

__global__ void featureresponseKernel(int *data, int *mask, int *new_mask, int samples, int feature, int threshold)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;
	if(mask[index]){
		if (data[(feature*samples) + index] > threshold)
			new_mask[index] = 1;
	}
}

void dfs(int features, int samples, int depth, int *mask, int max_depth = 10, int min_split = 100) {
	if (depth >= max_depth)
		return;
	for(int i = 0; i<100; i++)
		feature_threshold[i] = {rand()%features,rand()%256};
	for(int i = 0; i<100; i++){
		int *dev_mask,*new_mask;
		float *imp;
		hipMalloc((void**)&dev_mask, samples * sizeof(int));
		hipMalloc((void**)&new_mask, samples * sizeof(int));
		hipMemcpy(dev_mask, mask, samples * sizeof(int), hipMemcpyHostToDevice);
		hipMalloc((void**)&imp, sizeof(float));
		//kernel here
		featureresponseKernel(data, dev_mask, new_mask, samples, feature_threshold[i].first, feature_threshold[i].second);
		hipMemcpy(ft_impurity + i, imp, sizeof(float), hipMemcpyDeviceToHost);
		hipFree(imp);
		hipFree(dev_mask);
		hipFree(new_mask);
	}
	float *min_imp = std::min_element(ft_impurity,ft_impurity+100);
	std::pair<int, int> ft = *(feature_threshold+(min_imp - ft_impurity));
	int *dev_mask, *new_mask;
	float *imp;
	hipMalloc((void**)&dev_mask, samples * sizeof(int));
	hipMalloc((void**)&new_mask, samples * sizeof(int));
	hipMemcpy(dev_mask, mask, samples * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&imp, sizeof(float));
	//kernel here
	int *d1_mask = new int[samples], *d2_mask = new int[samples];
	hipMemcpy(d1_mask, new_mask, samples * sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0; i<samples; i++)
		d2_mask[i] = mask[i] - d1_mask[i];
	hipFree(imp);
	hipFree(dev_mask);
	dfs(features, samples, depth + 1, d1_mask, max_depth, min_split);
	dfs(features, samples, depth + 1, d2_mask, max_depth, min_split);
}

int main()
{
	srand(time(NULL));
	const int samples = 5, features = 28 * 28;
	data = new int[features*samples];
	target = new int[samples];
	mask = new int[samples];
	for (int i = 0; i < samples; i++)
		mask[i] = 1;

    // Add vectors in parallel.
    hipError_t cudaStatus = dftrain(data, features, samples);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t dftrain(const int *data, const int *target, unsigned int features, unsigned int samples)
{
    int *dev_data;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_data, features * samples * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_data, data, features * samples * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	dfs(features, samples, 0, mask);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

Error:
    hipFree(dev_data);
    
    return cudaStatus;
}
